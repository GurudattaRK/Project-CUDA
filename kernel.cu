#include "hip/hip_runtime.h"
#include<stdio.h>
#include<conio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define CIPHER_BLOCK 128

// CUDA kernel
__global__ void supCUDA(char* key,char* initaddr,int itersize, int rounds ) 
{
    int id = (threadIdx.x) + (blockIdx.x * 32);
    
    unsigned long long int  hostaddr =(unsigned long long int)initaddr ;

    if(threadIdx.x == 0)
    printf("%d iter = %d : %llu\n",id,itersize,hostaddr);

    hostaddr = hostaddr + (id*128);
    // unsigned long long int addr = hostaddr + (id*itersize) ;
    char* thread_addr = (char*)hostaddr;
    unsigned char tic=32;
    while (rounds >0)
    {
        if(threadIdx.x == 0)
        printf("tid = %d : %llu\n",id,(unsigned long long int*)thread_addr);
        for(int i=0 ; i<128 ; i++)
        {
            thread_addr[i] = thread_addr[i]^key[i];
        } 
        thread_addr = thread_addr+itersize;
        rounds--;
    }
    
    
}


int main()
{
    // remove("1.txt");
    FILE* reader = fopen("1.txt","rb");
    FILE* writer = fopen("2.txt","wb");

    int blocks=2,warp_size =32;
    int rounds =4;

    int iter_size = blocks*warp_size*CIPHER_BLOCK;
    unsigned long long int total_size = blocks*warp_size*CIPHER_BLOCK*rounds;

    char* data = (char*)calloc(1,total_size);
    // memset(data,69,total_size);
    char key[130] = "qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,wuriov\0\0";

    // Allocate memory on the device
    char *CudaData, *CudaKey;

    
    data = (char*)calloc(1,total_size);

    fread(data, total_size,1,reader);

    printf("1\n");
    hipMalloc((void**)&CudaData, total_size);
    // hipMalloc((void**)&deviceOutput, BUF_SIZE);
    hipMalloc((void**)&CudaKey, 129);
    printf("2\n");

    hipMemcpy(CudaData, data, total_size, hipMemcpyHostToDevice);
    hipMemcpy(CudaKey, key, 128, hipMemcpyHostToDevice);
    printf("3\n");

    supCUDA<<<2,32>>>(CudaKey,CudaData,iter_size,rounds);
    printf("4\n");

    hipDeviceSynchronize();
    printf("5\n");


    hipMemcpy(data, CudaData,  total_size, hipMemcpyDeviceToHost);
    printf("%32767s\n",data);
    printf("6\n");

    hipFree(CudaData);
    hipFree(CudaKey);


    fwrite(data, total_size,1,writer);
    // fwrite(data,1,32768,writer);

    
    free(data);

    fclose(writer);
    fclose(reader);
    printf("7\n");


    return 0;
}