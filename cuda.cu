#include "hip/hip_runtime.h"
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>

#define BUF_SIZE 1073741824

// CUDA kernel
__global__ void supCUDA(char* key, char* input) {
    int x = (threadIdx.x * 128) + (blockIdx.x * 32 * 128);
    for (int j = 0; j < 32768; j++) {
        for (int i = 0; i < 128; i++) {
            input[x + i] = input[x + i] ^ key[i];
        }
        x = x + 32768;
    }
}

int main() 
{
    
    // Allocate memory on the host for input and output data
    char* hostInput = (char*)malloc(BUF_SIZE);
    // char* hostOutput = (char*)malloc(BUF_SIZE);
    char key[130] = "qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,wuriov\0\0";
    
    clock_t begin,end;
    begin = clock();
    double time_spent;

    FILE* reader = fopen("1","rb");
    FILE* writer = fopen("1.CUDAlock","wb");

    fread(hostInput,BUF_SIZE,1,reader);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);
    unsigned long long int freemem,totalmem;
    hipMemGetInfo(&freemem,&totalmem);

    const char* cudaSourceCode = R"(
        extern "C" __global__ void supCUDA(char* key, char* input) {
            int x = (threadIdx.x * 128) + (blockIdx.x * 32 * 128);
            for (int j = 0; j < 32768; j++) {
                for (int i = 0; i < 128; i++) {
                    input[x + i] = input[x + i] ^ key[i];
                }
                x = x + 32768;
            }
        }
    )";

    // Initialize CUDA
    hipSetDevice(0);

    
    // Fill hostInput with data or load it from a file

    // Allocate memory on the device
    char* deviceInput;
    // char* deviceOutput;
    char* devicekey;
    hipMalloc((void**)&deviceInput, BUF_SIZE);
    // hipMalloc((void**)&deviceOutput, BUF_SIZE);
    hipMalloc((void**)&devicekey, 129);

    // Copy data from host to device
    hipMemcpy(deviceInput, hostInput, BUF_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(devicekey, key, 128, hipMemcpyHostToDevice);

    // Create an NVRTC program and compile the CUDA source code
    hiprtcProgram program;
    hiprtcCreateProgram(&program, cudaSourceCode, "cuda.cu", 0, NULL, NULL);
    const char* options[] = { NULL };
    hiprtcCompileProgram(program,0, options);

    // Retrieve and print the compilation log
    size_t logSize;
    hiprtcGetProgramLogSize(program, &logSize);
    char* log = (char*)malloc(logSize);
    hiprtcGetProgramLog(program, log);
    printf("Compilation log:\n%s\n", log);
    free(log);

    // Retrieve the PTX code
    size_t ptxSize;
    hiprtcGetCodeSize(program, &ptxSize);
    char* ptx = (char*)malloc(ptxSize);
    hiprtcGetCode(program, ptx);

    // Load the PTX code into a CUDA module
    hipModule_t cuModule;
    hipModuleLoadDataEx(&cuModule, ptx, 0, 0, 0);

    // Get a function from the module
    hipFunction_t cuFunction;
    hipModuleGetFunction(&cuFunction, cuModule, "supCUDA");

    // Launch the kernel
    dim3 blockDim(32, 1);
    dim3 gridDim(8, 1);
    void* kernelParams[] = { &devicekey, &deviceInput };
    hipModuleLaunchKernel(cuFunction, gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z, 0, 0, kernelParams, 0);
    hipCtxSynchronize();

    // Copy the results from the device to the host
    hipMemcpy(hostInput, deviceInput, BUF_SIZE, hipMemcpyDeviceToHost);

    // Perform any necessary post-processing with hostOutput data
    fwrite(hostInput,BUF_SIZE,1,writer);

    fclose(reader);
    fclose(writer);


    // Clean up resources
    hiprtcDestroyProgram(&program);
    free(ptx);
    free(hostInput);
    // free(hostOutput);
    hipFree(deviceInput);
    // hipFree(deviceOutput);
    hipFree(devicekey);
    hipModuleUnload(cuModule);

    end = clock();
    time_spent = (double)(end - begin) / CLOCKS_PER_SEC;    
    printf("\n\nExecution time: %f seconds ",time_spent);

    hipGetDeviceProperties(&deviceProp,0);
    // printf("\n%s",deviceProp.name);
    printf("\nProcessors: %d",deviceProp.multiProcessorCount);
    printf("\nCompute strength: %d",deviceProp.multiProcessorCount*deviceProp.warpSize);
    printf("\nFree memory: %llu\nTotal memory: %llu\n",freemem,totalmem);

    char file1[20]= "1.singlelock";
    char file2[20]="1.CUDAlock";
    // printf("Enter first file's name/path to file:\n");
    // scanf("%s",file1);
    // printf("Enter Second file's name/path to file:\n");
    // scanf("%s",file2);

    FILE* reader1 = fopen(file1,"rb");
    FILE* reader2 = fopen(file2,"rb");
    if(reader==NULL)
    {
        printf("\nError in opening/finding the file %s",file1);
        return 0;
    }
    if(reader2==NULL)
    {
        printf("\nError in opening/finding the file %s",file2);
        return 0;
    }
    unsigned long long int check=0,in[1],out[1];
    unsigned long long int count=0,size;

    fseek(reader,0,SEEK_END);
    size= ftell(reader);
    fseek(reader,0,SEEK_SET);

    printf("Checking files...\n");

    while (count<size)
    {
        fread(in,8,1,reader);
        fread(out,8,1,reader2);
        check = in[0]^out[0];
        if(check)
        {
            printf("\nERROR \n\nFound at byte %llu",count);
            goto jump;
        }
        count = count+8;
    }
    printf("\nNO ERROR\n\n%llu bytes verified",count);
    jump:
    fclose(reader);
    fclose(reader2);
    remove("1");
    remove("1.singlelock");
    remove("1.CUDAlock");
    
    return 0;
}
