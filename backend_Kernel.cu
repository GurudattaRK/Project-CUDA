#include "hip/hip_runtime.h"
#include<stdio.h>
#include<conio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void supCUDA(char* key,char* initaddr,int itersize, int rounds ) 
{
    int id = (threadIdx.x) + (blockIdx.x * 32);
    
    unsigned long long int  hostaddr =(unsigned long long int)initaddr ;

    hostaddr = hostaddr + (id*128);
    // unsigned long long int addr = hostaddr + (id*itersize) ;
    char* thread_addr = (char*)hostaddr;

    if(threadIdx.x == 0)
        printf("tid = %d : %llu\n",id,(unsigned long long int*)thread_addr);

    while (rounds >0)
    {
        //ENCRYPTION BEGINS HERE (128 bytes from thread_addr with 128 bytes of key)  
        for(int i=0 ; i<128 ; i++)
        {
            thread_addr[i] = thread_addr[i]^key[i];
        } 
        //ENCRYPTION ENDS HERE
        thread_addr = thread_addr+itersize;
        rounds--;
    }
    
}

void sequentiel(char* key,char* data,int rounds)
{
    // unsigned long long int end = rounds *128;
    while(rounds>0 || rounds !=0)
    {
        for(int i=0; i<128;i++)
        {
            data[i]= data[i] ^ key[i];
        }
        data = data +128;
        rounds -- ;
    }

}

int main()
{

    // char* argt =  argv[1];
    // char* argx = argv[2];
    // printf("\nArguement 1:%s\n",argt);
    // printf("Arguement 2:%s\n",argx);

    // hipDeviceProp_t deviceProp;
    // hipGetDeviceProperties(&deviceProp,0);

    // int x= 9;//atoi(argt);
    // int y= 3;//atoi(argx);

    // hipMemGetInfo(&free_bytes, &total_bytes);
    unsigned long long int temp,residue,cuda_malloc_size,filesize,iter_size,max_free,available_mem,residue_offset,kernel_rounds=0;
    long long int rounds=0;
    char *CudaData, *CudaKey;



    FILE *reader = fopen("1.txt","rb");
    FILE* writer = fopen("2.txt","wb");

    int blocks =4,threads=32;

    filesize = 128*128 + 128;
    available_mem = 128*128;
    char key[130] = "qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,qwertyuiopasdfghjklzxcvbnm,ouoq,re,xqe0,wuriov\0\0";



    hipMalloc((void**)&CudaKey, 129);
    hipMemcpy(CudaKey, key, 128, hipMemcpyHostToDevice);

    iter_size =blocks*threads*128 ;// (deviceProp.multiProcessorCount*deviceProp.warpSize*128);
    residue = available_mem % iter_size;
    max_free = available_mem-residue;
    // unsigned long long int total = total_bytes;

    printf("\nfree available:  %llu",available_mem);
    printf("\nmaxfree:  %llu",max_free);
    printf("\nfilesize:  %llu",filesize);
    printf("\n=====================\n");

    char *hostptr,*hostptrcpy;

    hostptr = (char*)malloc(filesize);
    fread(hostptr, filesize,1,reader);

    hostptrcpy = hostptr;

    residue = 0;

                                        printf("*1\n");

    if(filesize>max_free)
    {
        cuda_malloc_size=max_free;
        rounds = filesize / max_free;
        residue = filesize % max_free;
        residue_offset = rounds * max_free;
        kernel_rounds = max_free / iter_size;
        // printf("\nfull rounds: %lld\n",rounds);
                                        printf("*2\n");

        hipMalloc((void**)&CudaData, cuda_malloc_size);

        while(rounds>0 || rounds != 0)
        {
                                        printf("*3\n");

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();
                                        printf("*4\n");

            printf("Case1 heavy kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
                                        printf("*5\n");

            hostptrcpy = hostptrcpy + max_free;

            rounds--;
            printf("round:%llu\n",rounds);


        }
                                        printf("*6\n");


    }
    else
    {
        rounds=0; 
                                        printf("*7\n");

        if(filesize <= iter_size)
        {
            residue = filesize;
            residue_offset = 0;
            kernel_rounds = 0;
                                        printf("*8\n");

        }
        else
        {
                                        printf("*9\n");

            temp =  filesize/iter_size;
            cuda_malloc_size = temp * iter_size;
            residue = filesize - cuda_malloc_size;
            residue_offset = cuda_malloc_size;
            kernel_rounds = temp ;

            hipMalloc((void**)&CudaData, cuda_malloc_size);

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
                                        printf("*A\n");

            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();

            printf("\nCase2 mid kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
                                        printf("*B\n");

        }
        
    }

    printf("\n=====================\n");

    if(residue!=0)
    {
        printf("\nresidue:%lld\n",residue);

        if(residue >= iter_size*10)
        {
            temp =  residue/iter_size;
            cuda_malloc_size = temp * iter_size;
            residue = residue - cuda_malloc_size;
            residue_offset = cuda_malloc_size;
            kernel_rounds = temp ;
                                        printf("*C\n");

            //Launch CUDA kernel here
            hipMalloc((void**)&CudaData, cuda_malloc_size);

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();
            printf("\nCase3 mid kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
                                        printf("*D\n");

            hostptrcpy = hostptrcpy + residue_offset;
        }
        if(residue > 0 )
        {
                                        printf("*E\n");

            if((residue%128)==0)
            {
                rounds = residue / 128;

            }
            else
            {
                temp = residue%128 ;
                residue = residue + (128-temp);
                rounds = residue/128 ;
            }
            // printf("\nmid rounds:%lld",rounds);


            hostptrcpy= hostptr + residue_offset;

            
            printf("\nCase4 light sequential execution\n");
            printf("\nSequential rounds: %lld",rounds);
            
            sequentiel(key,hostptrcpy,rounds);
            //Launch normal kernel here
            printf("\n=====================\n");
            printf("\nresidue:%lld",residue);
        }

    }
    
    // hipMalloc((void**)&cudaptr,cuda_malloc_size);


    // hipMemcpy(cudaptr,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);

    // printf("\n===============\n1:  %llu",available_mem);
    // printf("\n2:  %llu\n",max_free);
    // printf("3:  %llu\n",max_free/iter_size);

    // supCUDA<<<4,1024>>>(x, y);
    // hipDeviceSynchronize();
    // hipMemcpy(hostptrcpy,cudaptr,cuda_malloc_size,hipMemcpyDeviceToDevice);


    // hipFree(cudaptr);
    
    hipFree(CudaData);
    hipFree(CudaKey);

    fwrite(hostptr, filesize,1,writer);

    fclose(writer);
    fclose(reader);

    free(hostptr);
    // // printf("\n%s",deviceProp.name);
    // printf("\nProcessors: %d",deviceProp.multiProcessorCount);
    // printf("\nWarp: %d",deviceProp.warpSize);
    // printf("\nIteration size: %d",deviceProp.multiProcessorCount*deviceProp.warpSize*128);

    return 0;
}