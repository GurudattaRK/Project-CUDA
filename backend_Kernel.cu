#include "hip/hip_runtime.h"
#include<stdio.h>
#include<conio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void supCUDA(char* key,char* initaddr,int itersize, int rounds ) 
{
    int id = (threadIdx.x) + (blockIdx.x * 32);
    
    unsigned long long int  hostaddr =(unsigned long long int)initaddr ;

    hostaddr = hostaddr + (id*128);
    // unsigned long long int addr = hostaddr + (id*itersize) ;
    char* thread_addr = (char*)hostaddr;

    if(threadIdx.x == 0)
        printf("tid = %d : %llu\n",id,(unsigned long long int*)thread_addr);

    while (rounds >0)
    {
        //ENCRYPTION BEGINS HERE (128 bytes from thread_addr with 128 bytes of key)  
        for(int i=0 ; i<128 ; i++)
        {
            thread_addr[i] = thread_addr[i]^key[i];
        } 
        //ENCRYPTION ENDS HERE
        thread_addr = thread_addr+itersize;
        rounds--;
    }
    
}

void sequentiel(char* key,char* data,int rounds)
{
    // unsigned long long int end = rounds *128;
    while(rounds>0 || rounds !=0)
    {
        for(int i=0; i<128;i++)
        {
            data[i]= data[i] ^ key[i];
        }
        data = data +128;
        rounds -- ;
    }

}

int main(int argc, char *argv[])
{

    char* argt =  argv[1];
    printf("\nArguement 1:%s\n",argt);
    FILE* tempfile = fopen(argt,"r");
    char line[257];
    char *lines[5]; // Array to store pointers to each line string
    int line_count = 0;

    // Read and store lines
    while (fgets(line, 257, tempfile) != NULL && line_count < 5) 
    {
        // Remove the newline character at the end of the line
        line[strcspn(line, "\n")] = '\0';

        // Skip empty lines
        if (strlen(line) == 0) {
            continue;
        }

        // Allocate memory for the line string and copy the line
        lines[line_count] = (char*)malloc((strlen(line) + 1) * sizeof(char));
        strcpy(lines[line_count], line);
        line_count++;
    }

    fclose(tempfile);

    //  char* argt =  argv[1];
    // char* argx = argv[2];
    // printf("\nArguement 1:%s\n",argt);
    // printf("Arguement 2:%s\n",argx);

    // hipDeviceProp_t deviceProp;
    // hipGetDeviceProperties(&deviceProp,0);

    // int x= 9;//atoi(argt);
    // int y= 3;//atoi(argx);

    // hipMemGetInfo(&free_bytes, &total_bytes);

    unsigned long long int temp,residue,cuda_malloc_size,filesize,iter_size,max_free,available_mem,residue_offset,kernel_rounds=0;
    long long int rounds=0;
    char *CudaData, *CudaKey ,*inputfile, *outputfile,*size,*mode,*random;

    inputfile = lines[1];
    outputfile = lines[2];
    size = lines[3];
    mode = lines[4];

    FILE *reader = fopen(inputfile,"rb");
    FILE* writer = fopen(outputfile,"wb");

    int blocks =4,threads=32;

    filesize  = strtoull(size,&random,10);
    
    available_mem = 128*128;
    char *key = lines[0] ;

    printf("inputfile :%s\n",inputfile);
    printf("outputfile :%s\n",outputfile);
    printf("key :%s\n",key);
    printf("size :%llu\n",size);
    printf("mode :%s\n",mode);



    hipMalloc((void**)&CudaKey, 129);
    hipMemcpy(CudaKey, key, 128, hipMemcpyHostToDevice);

    iter_size =blocks*threads*128 ;// (deviceProp.multiProcessorCount*deviceProp.warpSize*128);
    residue = available_mem % iter_size;
    max_free = available_mem-residue;
    // unsigned long long int total = total_bytes;

    printf("\nfree available:  %llu",available_mem);
    printf("\nmaxfree:  %llu",max_free);
    printf("\nfilesize:  %llu",filesize);
    printf("\n=====================\n");

    char *hostptr,*hostptrcpy;

    hostptr = (char*)malloc(filesize);
    fread(hostptr, filesize,1,reader);

    hostptrcpy = hostptr;

    residue = 0;

                                        printf("*1\n");

    if(filesize>max_free)
    {
        cuda_malloc_size=max_free;
        rounds = filesize / max_free;
        residue = filesize % max_free;
        residue_offset = rounds * max_free;
        kernel_rounds = max_free / iter_size;
        // printf("\nfull rounds: %lld\n",rounds);
                                        printf("*2\n");

        hipMalloc((void**)&CudaData, cuda_malloc_size);

        while(rounds>0 || rounds != 0)
        {
                                        printf("*3\n");

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();
                                        printf("*4\n");

            printf("Case1 heavy kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
                                        printf("*5\n");

            hostptrcpy = hostptrcpy + max_free;

            rounds--;
            printf("round:%llu\n",rounds);


        }
                                        printf("*6\n");


    }
    else
    {
        rounds=0; 
                                        printf("*7\n");

        if(filesize <= iter_size)
        {
            residue = filesize;
            residue_offset = 0;
            kernel_rounds = 0;
                                        printf("*8\n");

        }
        else
        {
                                        printf("*9\n");

            temp =  filesize/iter_size;
            cuda_malloc_size = temp * iter_size;
            residue = filesize - cuda_malloc_size;
            residue_offset = cuda_malloc_size;
            kernel_rounds = temp ;

            hipMalloc((void**)&CudaData, cuda_malloc_size);

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
                                        printf("*A\n");

            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();

            printf("\nCase2 mid kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
                                        printf("*B\n");

        }
        
    }

    printf("\n=====================\n");

    if(residue!=0)
    {
        printf("\nresidue:%lld\n",residue);

        if(residue >= iter_size*10)
        {
            temp =  residue/iter_size;
            cuda_malloc_size = temp * iter_size;
            residue = residue - cuda_malloc_size;
            residue_offset = cuda_malloc_size;
            kernel_rounds = temp ;
                                        printf("*C\n");

            //Launch CUDA kernel here
            hipMalloc((void**)&CudaData, cuda_malloc_size);

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();
            printf("\nCase3 mid kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
                                        printf("*D\n");

            hostptrcpy = hostptrcpy + residue_offset;
        }
        if(residue > 0 )
        {
                                        printf("*E\n");

            if((residue%128)==0)
            {
                rounds = residue / 128;

            }
            else
            {
                temp = residue%128 ;
                residue = residue + (128-temp);
                rounds = residue/128 ;
            }
            // printf("\nmid rounds:%lld",rounds);


            hostptrcpy= hostptr + residue_offset;

            
            printf("\nCase4 light sequential execution\n");
            printf("\nSequential rounds: %lld",rounds);
            
            sequentiel(key,hostptrcpy,rounds);
            //Launch normal kernel here
            printf("\n=====================\n");
            printf("\nresidue:%lld",residue);
        }

    }
    
    // hipMalloc((void**)&cudaptr,cuda_malloc_size);


    // hipMemcpy(cudaptr,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);

    // printf("\n===============\n1:  %llu",available_mem);
    // printf("\n2:  %llu\n",max_free);
    // printf("3:  %llu\n",max_free/iter_size);

    // supCUDA<<<4,1024>>>(x, y);
    // hipDeviceSynchronize();
    // hipMemcpy(hostptrcpy,cudaptr,cuda_malloc_size,hipMemcpyDeviceToDevice);


    // hipFree(cudaptr);
    
    hipFree(CudaData);
    hipFree(CudaKey);

    fwrite(hostptr, filesize,1,writer);

    fclose(writer);
    fclose(reader);

    for (int i = 0; i < line_count; i++) 
    {
        printf("Line %d: %s\n", i, lines[i]);
        free(lines[i]);
    }

    free(hostptr);
    // // printf("\n%s",deviceProp.name);
    // printf("\nProcessors: %d",deviceProp.multiProcessorCount);
    // printf("\nWarp: %d",deviceProp.warpSize);
    // printf("\nIteration size: %d",deviceProp.multiProcessorCount*deviceProp.warpSize*128);

    return 0;
}