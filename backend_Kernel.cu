#include "hip/hip_runtime.h"
#include<stdio.h>
#include<conio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void supCUDA(char* key,char* initaddr,int itersize, int rounds ) 
{
    int id = (threadIdx.x) + (blockIdx.x * 32);
    
    unsigned long long int  hostaddr =(unsigned long long int)initaddr ;

    hostaddr = hostaddr + (id*128);
    // unsigned long long int addr = hostaddr + (id*itersize) ;
    char* thread_addr = (char*)hostaddr;

    // if(threadIdx.x == 0 && blockIdx.x == 0)4
    //     printf("tid = %d : %llu\n",id,(unsigned long long int*)thread_addr);

    while (rounds >0)
    {
        //ENCRYPTION BEGINS HERE (128 bytes from thread_addr with 128 bytes of key)  
        for(int i=0 ; i<128 ; i++)
        {
            thread_addr[i] = thread_addr[i]^key[i];
        } 
        //ENCRYPTION ENDS HERE
        thread_addr = thread_addr+itersize;
        rounds--;
    }
    
}

void sequentiel(char* key,char* data,int rounds)
{
    // unsigned long long int end = rounds *128;
    while(rounds>0 || rounds !=0)
    {
        for(int i=0; i<128;i++)
        {
            data[i]= data[i] ^ key[i];
        }
        data = data +128;
        rounds -- ;
    }

}

int main(int argc, char *argv[])
{

    char* argt =  argv[1];
    printf("\nArguement 1:%s\n",argt);
    FILE* tempfile = fopen(argt,"r");
    char line[257];
    char *lines[5]; // Array to store pointers to each line string
    int line_count = 0;

    // Read and store lines
    while (fgets(line, 257, tempfile) != NULL && line_count < 5) 
    {
        // Remove the newline character at the end of the line
        line[strcspn(line, "\n")] = '\0';

        // Skip empty lines
        if (strlen(line) == 0) {
            continue;
        }

        // Allocate memory for the line string and copy the line
        lines[line_count] = (char*)malloc((strlen(line) + 1) * sizeof(char));
        strcpy(lines[line_count], line);
        line_count++;
    }

    fclose(tempfile);


    unsigned long long int temp,residue,cuda_malloc_size,filesize,iter_size,max_free,available_mem,residue_offset,kernel_rounds=0;
    long long int rounds=0;
    char *CudaData, *CudaKey ,*inputfile, *outputfile,*size,*mode,*random;

    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

    inputfile = lines[1];
    outputfile = lines[2];
    size = lines[3];
    mode = lines[4];

    FILE *reader = fopen(inputfile,"rb");
    FILE* writer = fopen(outputfile,"wb");

    int blocks =4,threads=32;
    blocks = deviceProp.multiProcessorCount;
    threads =deviceProp.warpSize;

    filesize  = strtoull(size,&random,10);
    
    hipMemGetInfo(&available_mem,&temp);
    char *key = lines[0] ;

    printf("inputfile :%s\n",inputfile);
    printf("outputfile :%s\n",outputfile);
    printf("key :%s\n",key);
    printf("size :%llu\n",filesize);
    printf("mode :%s\n",mode);
    printf("============================\n");


    hipMalloc((void**)&CudaKey, 129);
    hipMemcpy(CudaKey, key, 128, hipMemcpyHostToDevice);

    iter_size =blocks*threads*128 ;// (deviceProp.multiProcessorCount*deviceProp.warpSize*128);
    residue = available_mem % iter_size;
    max_free = available_mem-residue;
    // unsigned long long int total = total_bytes;

    printf("\nfree mem:  %llu",available_mem);
    printf("\nmaxfree:  %llu",max_free);
    printf("\nitersize:  %llu",iter_size);
    printf("\nprocessors:  %d",blocks);
    printf("\nwarp size:  %d",threads);
    printf("\n=====================\n");

    char *hostptr,*hostptrcpy;

    hostptr = (char*)malloc(filesize);
    fread(hostptr, filesize,1,reader);

    hostptrcpy = hostptr;

    residue = 0;

                                        printf("*1\n");

    if(filesize>max_free)
    {
        cuda_malloc_size=max_free;
        rounds = filesize / max_free;
        residue = filesize % max_free;
        residue_offset = rounds * max_free;
        kernel_rounds = max_free / iter_size;
        // printf("\nfull rounds: %lld\n",rounds);
                                        printf("*2\n");

        hipMalloc((void**)&CudaData, cuda_malloc_size);

        while(rounds>0 || rounds != 0)
        {
                                        printf("*3\n");

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();
                                        printf("*4\n");

            printf("Case1 heavy kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
                                        printf("*5\n");

            hostptrcpy = hostptrcpy + max_free;

            rounds--;
            printf("round:%llu\n",rounds);


        }
        hipFree(CudaData);

                                        printf("*6\n");


    }
    else
    {
        rounds=0; 
                                        printf("*7\n");

        if(filesize <= iter_size)
        {
            residue = filesize;
            residue_offset = 0;
            kernel_rounds = 0;
                                        printf("*8\n");

        }
        else
        {
                                        printf("*9\n");

            temp =  filesize/iter_size;
            cuda_malloc_size = temp * iter_size;
            residue = filesize - cuda_malloc_size;
            residue_offset = cuda_malloc_size;
            kernel_rounds = temp ;

            hipMalloc((void**)&CudaData, cuda_malloc_size);

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
                                        printf("*A\n");

            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();

            printf("Case2 mid kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
            hipFree(CudaData);
                                        printf("*B\n");

        }
        
    }

    printf("\n=====================\n");

    if(residue!=0)
    {
        printf("\nresidue:%lld\n",residue);

        if(residue >= iter_size*10)
        {
            temp =  residue/iter_size;
            cuda_malloc_size = temp * iter_size;
            residue = residue - cuda_malloc_size;
            residue_offset = cuda_malloc_size;
            kernel_rounds = temp ;
                                        printf("*C\n");

            //Launch CUDA kernel here
            hipMalloc((void**)&CudaData, cuda_malloc_size);

            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);
            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();
            printf("Case3 mid kernel launched\n");
            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
                                        printf("*D\n");

            hipFree(CudaData);
            hostptrcpy = hostptrcpy + residue_offset;
        }
        if(residue > 0 )
        {
                                        printf("*E\n");

            if((residue%128)==0)
            {
                rounds = residue / 128;

            }
            else
            {
                temp = residue%128 ;
                residue = residue + (128-temp);
                rounds = residue/128 ;
            }
            // printf("\nmid rounds:%lld",rounds);


            hostptrcpy= hostptr + residue_offset;

            printf("\n=====================\n");
            printf("residue:%lld\n",residue);
            printf("Case4 light sequential execution\n");
            printf("Sequential rounds: %lld\n",rounds);
            
            sequentiel(key,hostptrcpy,rounds);
            //Launch normal kernel here
            
        }

    }
    
    // hipMalloc((void**)&cudaptr,cuda_malloc_size);


    // hipMemcpy(cudaptr,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);

    // printf("\n===============\n1:  %llu",available_mem);
    // printf("\n2:  %llu\n",max_free);
    // printf("3:  %llu\n",max_free/iter_size);

    // supCUDA<<<4,1024>>>(x, y);
    // hipDeviceSynchronize();
    // hipMemcpy(hostptrcpy,cudaptr,cuda_malloc_size,hipMemcpyDeviceToDevice);


    // hipFree(cudaptr);
    
    // hipFree(CudaData);
    hipFree(CudaKey);

    fwrite(hostptr, filesize,1,writer);

    fclose(writer);
    fclose(reader);

    for (int i = 0; i < line_count; i++) 
    {
        // printf("Line %d: %s\n", i, lines[i]);
        free(lines[i]);
    }

    free(hostptr);
    // // printf("\n%s",deviceProp.name);
    // printf("\nProcessors: %d",deviceProp.multiProcessorCount);
    // printf("\nWarp: %d",deviceProp.warpSize);
    // printf("\nIteration size: %d",deviceProp.multiProcessorCount*deviceProp.warpSize*128);

    return 0;
}
